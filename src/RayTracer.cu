#include "hip/hip_runtime.h"
#include <limits>
#include "RayTracer.h"
#include "Image.h"
#include "Object.h"
#include "Sphere.h"
#include "Intersection.h"
#include "Light.h"

using namespace std;

#define ERROR_HANDLER(x) ErrorHandler(x, __FILE__, __LINE__)
#define TILE_WIDTH 32

static void ErrorHandler(hipError_t err, const char *file, int line) {
   if (err != hipSuccess) {
      fprintf(stderr, "%s in line %d: %s\n", file, line, hipGetErrorString(err));
      exit(EXIT_FAILURE);
   }
}

RayTracer::RayTracer(int width_, int height_, int maxReflections_, int superSamples_,
 int depthComplexity_) : width(width_), height(height_),
 maxReflections(maxReflections_), superSamples(superSamples_), camera(Camera()),
 imageScale(1), depthComplexity(depthComplexity_), dispersion(5.0f), raysCast(0) {}

RayTracer::~RayTracer() {
}

void RayTracer::traceRays(string fileName) {
   Color* devImage;
   Sphere* devSpheres;
   Light* devLights;
   RayTracer* devRayTracer;
   int columnsCompleted = 0;
   Image image(width, height);

   // Reset depthComplexity to avoid unnecessary loops.
   if (dispersion < 0) {
      depthComplexity = 1;
   }

   ERROR_HANDLER(hipMalloc((void**)&devSpheres, spheres.size() * sizeof(Sphere)));
   ERROR_HANDLER(hipMemcpy(devSpheres, &spheres.front(),
    spheres.size() * sizeof(Sphere), hipMemcpyHostToDevice));

   ERROR_HANDLER(hipMalloc((void**)&devLights, lights.size() * sizeof(Light)));
   ERROR_HANDLER(hipMemcpy(devLights, &lights.front(),
    lights.size() * sizeof(Light), hipMemcpyHostToDevice));

   ERROR_HANDLER(hipMalloc((void**)&devImage, width * height * sizeof(Color)));

   ERROR_HANDLER(hipMalloc((void**)&devRayTracer, sizeof(RayTracer)));
   ERROR_HANDLER(hipMemcpy(devRayTracer, this, sizeof(RayTracer), hipMemcpyHostToDevice));

   int gridWidth = ceil((float)width/TILE_WIDTH);
   int gridHeight = ceil((float)height/TILE_WIDTH);

   dim3 dimGrid(gridWidth, gridHeight);
   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

   cudaTraceRays<<<dimGrid, dimBlock>>>(devSpheres, devLights, devImage,
    devRayTracer);

   ERROR_HANDLER(hipMemcpy(image.getPixmap(), devImage,
    width * height * sizeof(Color), hipMemcpyDeviceToHost));

   ERROR_HANDLER(hipFree(devImage));
   ERROR_HANDLER(hipFree(devSpheres));
   ERROR_HANDLER(hipFree(devLights));
   ERROR_HANDLER(hipFree(devRayTracer));

   image.WriteTga(fileName.c_str(), false);
}

__global__ void cudaTraceRays(Sphere* spheres, Light* lights,
 Color* image, RayTracer* rayTracer) {
   int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
   int y = blockIdx.y * TILE_WIDTH + threadIdx.y;

   if (x < rayTracer->width && y < rayTracer->height) {
      Color* color = image + (x * rayTracer->height + y);
      color->r = 0.5;
      color->g = 0.5;
      color->b = 0.5;
   }

   //for (int x = 0; x < width; x++) {
   //   // Update percent complete.
   //   columnsCompleted++;
   //   float percentage = columnsCompleted/(float)width * 100;
   //   cout << '\r' << (int)percentage << '%';
   //   fflush(stdout);

   //   for (int y = 0; y < height; y++) {
   //      image.pixel(x, y, castRayForPixel(x, y));
   //   }
   //}

   //cout << "\rDone!" << endl;
   //cout << "Rays cast: " << raysCast << endl;
}

Color RayTracer::castRayForPixel(int x, int y) {
   double rayX = (x - width / 2)/2.0;
   double rayY = (y - height / 2)/2.0;
   double pixelWidth = rayX - (x + 1 - width / 2)/2.0;
   double sampleWidth = pixelWidth / superSamples;
   double sampleStartX = rayX - pixelWidth/2.0;
   double sampleStartY = rayY - pixelWidth/2.0;
   double sampleWeight = 1.0 / (superSamples * superSamples);
   Color color;

   for (int x = 0; x < superSamples; x++) {
      for (int y = 0; y < superSamples; y++) {
         Vector imagePlanePoint = camera.lookAt -
          (camera.u * (sampleStartX + (x * sampleWidth)) * imageScale) +
          (camera.v * (sampleStartY + (y * sampleWidth)) * imageScale);

         color = color + (castRayAtPoint(imagePlanePoint) * sampleWeight);
      }
   }

   return color;
}

Color RayTracer::castRayAtPoint(Vector point) {
   Color color;

   for (int i = 0; i < depthComplexity; i++) {
      Ray viewRay(camera.position, point - camera.position, maxReflections);

      if (depthComplexity > 1) {
         Vector disturbance(
          (dispersion / RAND_MAX) * (1.0f * rand()),
          (dispersion / RAND_MAX) * (1.0f * rand()),
          0.0f);

         viewRay.origin = viewRay.origin + disturbance;
         viewRay.direction = point - viewRay.origin;
         viewRay.direction = viewRay.direction.normalize();
      }

      color = color + (castRay(viewRay) * (1 / (float)depthComplexity));
   }

   return color;
}

Color RayTracer::castRay(Ray ray) {
   raysCast++;
   Intersection intersection = getClosestIntersection(ray);

   if (intersection.didIntersect) {
      return performLighting(intersection);
   } else {
      return Color();
   }
}

Intersection RayTracer::getClosestIntersection(Ray ray) {
   Intersection closestIntersection(false);
   closestIntersection.distance = numeric_limits<double>::max();

   for (vector<Sphere>::iterator itr = spheres.begin(); itr < spheres.end(); itr++) {
      Intersection intersection = (*itr).intersect(ray);

      if (intersection.didIntersect && intersection.distance <
       closestIntersection.distance) {
         closestIntersection = intersection;
      }
   }

   return closestIntersection;
}

Color RayTracer::performLighting(Intersection intersection) {
   Color ambientColor = getAmbientLighting(intersection);
   Color diffuseAndSpecularColor = getDiffuseAndSpecularLighting(intersection);
   Color reflectedColor = getReflectiveLighting(intersection);

   return ambientColor + diffuseAndSpecularColor + reflectedColor;
}

Color RayTracer::getAmbientLighting(Intersection intersection) {
   return intersection.color * 0.2;
}

Color RayTracer::getDiffuseAndSpecularLighting(Intersection intersection) {
   Color diffuseColor(0.0, 0.0, 0.0);
   Color specularColor(0.0, 0.0, 0.0);

   for (vector<Light>::iterator itr = lights.begin(); itr < lights.end(); itr++) {
      Light* light = &(*itr);
      Vector lightOffset = light->position - intersection.intersection;
      double lightDistance = lightOffset.length();
      /**
       * TODO: Be careful about normalizing lightOffset too.
       */
      Vector lightDirection = lightOffset.normalize();
      double dotProduct = intersection.normal.dot(lightDirection);

      /**
       * Intersection is facing light.
       */
      if (dotProduct >= 0.0f) {
         Ray shadowRay = Ray(intersection.intersection, lightDirection, 1);
         Intersection shadowIntersection = getClosestIntersection(shadowRay);

         if (shadowIntersection.didIntersect &&
          shadowIntersection.distance < lightDistance) {
            /**
             * Position is in shadow of another object - continue with other lights.
             */
            continue;
         }

         diffuseColor = (diffuseColor + (intersection.color * dotProduct)) *
          light->intensity;
         specularColor = specularColor + getSpecularLighting(intersection, light);
      }
   }

   return diffuseColor + specularColor;
}

Color RayTracer::getSpecularLighting(Intersection intersection, Light* light) {
   Color specularColor(0.0, 0.0, 0.0);
   double shininess = intersection.object->getShininess();

   if (shininess == NOT_SHINY) {
      /* Don't perform specular lighting on non shiny objects. */
      return specularColor;
   }

   Vector view = (intersection.ray.origin - intersection.intersection).normalize();
   Vector lightOffset = light->position - intersection.intersection;
   Vector reflected = reflectVector(lightOffset.normalize(), intersection.normal);

   double dot = view.dot(reflected);

   if (dot <= 0) {
      return specularColor;
   }

   double specularAmount = pow(dot, shininess) * light->intensity;

   specularColor.r = specularAmount;
   specularColor.g = specularAmount;
   specularColor.b = specularAmount;

   return specularColor;
}

Color RayTracer::getReflectiveLighting(Intersection intersection) {
   double reflectivity = intersection.object->getReflectivity();
   int reflectionsRemaining = intersection.ray.reflectionsRemaining;

   if (reflectivity == NOT_REFLECTIVE || reflectionsRemaining <= 0) {
      return Color();
   } else {
      Vector reflected = reflectVector(intersection.ray.origin, intersection.normal);
      Ray reflectedRay(intersection.intersection, reflected, reflectionsRemaining - 1);

      return castRay(reflectedRay) * reflectivity;
   }
}

Vector RayTracer::reflectVector(Vector vector, Vector normal) {
   return normal * 2 * vector.dot(normal) - vector;
}

void RayTracer::readScene(istream& in) {
   string type;

   in >> type;

   while (in.good()) {
      if (type[0] == '#') {
         // Ignore comment lines.
         getline(in, type);
      } else if (type.compare("sphere") == 0) {
         Vector center;
         double radius;
         Color color;
         double shininess;
         double reflectivity;

         in >> center.x >> center.y >> center.z;
         in >> radius;
         in >> color.r >> color.g >> color.b;
         in >> shininess;
         in >> reflectivity;

         addObject(new Sphere(center, radius, color, shininess, reflectivity));
      } else if (type.compare("light") == 0) {
         Vector position;
         double intensity;

         in >> position.x >> position.y >> position.z;
         in >> intensity;

         addLight(new Light(position, intensity));
      } else if (type.compare("dispersion") == 0) {
         in >> dispersion;
      } else if (type.compare("maxReflections") == 0) {
         in >> maxReflections;
      } else if (type.compare("cameraUp") == 0) {
         in >> camera.up.x;
         in >> camera.up.y;
         in >> camera.up.z;
      } else if (type.compare("cameraPosition") == 0) {
         in >> camera.position.x;
         in >> camera.position.y;
         in >> camera.position.z;
      } else if (type.compare("cameraLookAt") == 0) {
         in >> camera.lookAt.x;
         in >> camera.lookAt.y;
         in >> camera.lookAt.z;
      } else if (type.compare("imageScale") == 0) {
         in >> imageScale;
      } else {
         cerr << "Type not found: " << type << endl;
         exit(EXIT_FAILURE);
      }

      in >> type;
   }
}

